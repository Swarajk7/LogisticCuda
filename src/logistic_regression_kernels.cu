
#include <hip/hip_runtime.h>
//Put the kernel codes here.
//Optimizations:
//Weights in memory, shared memory, constant memory.
//Instead of using value, directly use intermediate_vector[i].
//Use hardware math functions for exp.

__global__ void memory_coalescedKernel(float *weights, float *X, float *y, float *intermediate_vector, int size, int N, int num_features)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = 0;
	float value = 0;
	//Start needs to be verified
	int start = index;
	//int start = index * num_features;
	if (start < N)
	{
		for (int i = 0; i < num_features; i++)
		{
			value += weights[i] * X[start + stride];
			stride += size;
		}
		value = 1 / (1 + expf(-value));
		//value = exp(value) / (1 + exp(value));
		value -= y[index];
		intermediate_vector[index] = value;
	}
}

__global__ void externalKernel(float * weights, float *grad_weights, float *X, float *intermediate_vector, int size, int N, const int num_features, const int X_dim, float learning_rate)
{
	__shared__ float values[32][29];
	__shared__ float intermediate_shared[32];
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	values[tx][ty] = 0.0f;
	__syncthreads();
	for (int m = 0; m < ceilf((N * 1.0f) / X_dim); m++)
	{
		int Col = tx + m * X_dim;
		int Row = ty;
		if (Col < N)
		{
			if (ty == 0)
				intermediate_shared[tx] = intermediate_vector[tx + m * X_dim];
			__syncthreads();

			values[tx][ty] += X[Row * size + Col] * intermediate_shared[tx];
		}
		__syncthreads();
	}
	if (tx == 0)
	{
		for (int q = 1; q < X_dim; q++)
		{
			values[tx][ty] += values[tx + q][ty];
		}
		grad_weights[ty] = values[tx][ty];
		//printf("Updating weight %f %d",grad_weights[ty], ty);
		weights[ty] -= ((learning_rate*grad_weights[ty])/N);	
	}
}

__global__ void uncoalescedKernel(float *weights, float *X, float *y, float *intermediate_vector, int size, int N, int num_features)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float value = 0;
	int start = index * num_features;
	if (start < N)
	{
		for (int i = 0; i < num_features; i++)
		{
			value += weights[i] * X[start + i];
		}
		value = exp(value) / (1 + exp(value));
		value -= y[index];
		intermediate_vector[index] = value;
	}
}


__global__ void evaluate_model(float *weights, float *X, float *y, float *intermediate_vector, int size, int N, int num_features, float * correct_val)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = 0;
	float value = 0;
	//Start needs to be verified
	int start = index;
	//int start = index * num_features;
	if (start < N)
	{
		for (int i = 0; i < num_features; i++)
		{
			value += weights[i] * X[start + stride];
			stride += size;
		}
		value = 1 / (1 + expf(-value));
		float y_pred;
		if(value>0.5)
			y_pred = 1.0f;
		else
			y_pred = 0.0f;
		if(y_pred == y[index])
			atomicAdd(correct_val,1);
	}
}


__global__ void printKernel(float * weights,float * inter_vector,int num_features){
	printf("WEIGHTS\n");
	for(int i=0;i<num_features;i++)
		printf("%f ",weights[i]);

	printf("Inter Values\n");
	for(int i=0;i<num_features;i++)
		printf("%f ",inter_vector[i]);
}