#include "hip/hip_runtime.h"
//Put the kernel codes here.
//Optimizations:
//Weights in memory, shared memory, constant memory.
//Instead of using value, directly use intermediate_vector[i].
//Use hardware math functions for exp.

__global__ void memory_coalescedKernel(float *weights, float *X, float *y, float *intermediate_vector, int size, int N, int num_features)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = 0;
	float value = 0;
	//Start needs to be verified
	int start = index
	//int start = index * num_features;
	if (start < N)
	{
		for (int i; i < num_features; i++)
		{
			value += weights[i] * X[start + stride];
			stride += size;
		}
		value = exp(value) / (1 + exp(value));
		value -= y[index];
		intermediate_vector[index] = value;
	}
}

__global__ void externalKernel(float *grad_weights, float *X, float *intermediate_vector, int size, int N, int num_features, int X_dim)
{
	__shared__ float values[X_dim][num_features] = 0;
	__shared__ float intermediate_shared[X_dim];
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	for (int m = 0; m < ceilf((N * 1.0f) / X_dim); m++)
	{
		Col = tx + m * X_dim;
		Row = ty;
		if (Col < N)
		{
			if (ty == 0)
				intermediate_shared[tx] = intermediate_vector[tx + m * X_dim];
			__syncthreads();

			values[tx][ty] += X[Row * size + Col] * intermediate_shared[tx];

			__syncthreads();
		}
	}
	if (tx == 0)
	{
		for (int q = 1; q < X_dim; q++)
		{
			values[tx][ty] += values[tx + q][ty];
		}
		grad_weights[ty] = values[tx][ty]
	}
}

__global__ void uncoalescedKernel(float *weights, float *X, float *y, float *intermediate_vector, int size, int N, int num_features)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float value = 0;
	int start = index * num_features;
	if (start < N)
	{
		for (int i; i < num_features; i++)
		{
			value += weights[i] * X[start + i];
		}
		value = exp(value) / (1 + exp(value));
		value -= y[index];
		intermediate_vector[index] = value;
	}
}